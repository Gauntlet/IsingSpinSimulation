#include "DataStructures.h"

using namespace kspace;

template <class elem_type>
MatrixShared::initialize( const uint32_t numofcols, const uint32_t numofrows )
{
	host = new Matrix( numofcols, numofrows, MemoryLocation::host );
	intermediary = new Matrix( numofcols, numofrows, MemoryLocation::device );
	HANDLE_ERROR( hipMalloc((void**)&device, sizeof(Matrix)) );
	HANDLE_ERROR( hipMemcpy(device, intermediary, sizeof(Matrix), hipMemcpyHostToDevice) );
	

	HANDLE_ERROR( hipMalloc(intermediary->_memloc, host->_memloc, sizeof(MemoryLocation), hipMemcpyHostToDevice);
	HANDLE_ERROR( hipMalloc(intermediary->_data, host->_data, sizeof(elem_type) * host->length(), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMalloc(intermediary->_length, host->_length, sizeof(uint32_t), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMalloc(intermediary->_numOfCols, host->_numOfCols, sizeof(uint32_t), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMalloc(intermediary->_numOfRows, host->_numOfRows, sizeof(uint32_t), hipMemcpyHostToDevice) );
}

template <class elem_type>
MatrixShared::MatrixShared( const uint32_t N )
{
	initialize( N, N );
}

template <class elem_type>
MatrixShared::MatrixShared( const uint32_t numofcols, const uint32_t numofrows )
{
	initialize( numofcols, numofrows );
}

template <class elem_type>
MatrixShared::~MatrixShared()
{
	HANDLE_ERROR( hipFree(device) );
	delete[] intermediary;
	delete[] host;
}

template <class elem_type>
void MatrixShared::host2device()
{
	HANDLE_ERROR( hipMalloc(intermediary->_data, host->_data, sizeof(elem_type) * host->length(), hipMemcpyHostToDevice) );
}

template <class elem_type>
void MatrixShared::device2host()
{
	HANDLE_ERROR( hipMalloc(host->_data, intermediary->_data, sizeof(elem_type) * host->length(), hipMemcpyDeviceToHost) );
}