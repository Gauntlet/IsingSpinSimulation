#include "GraphShared.h"
#include <string>
using namespace kspace::GRAPH;

GraphShared::GraphShared(const std::string filename)
{
	host_ptr = new Graph( filename, MemoryLocation::host );
	intermediary_ptr = new Graph( filename, MemoryLocation::device );
	hipMemcpy( device_ptr, intermediary_ptr, sizeof( Graph ), hipMemcpyHostToDevice );
}

GraphShared::~GraphShared()
{
	if ( nullptr != device )
	{
		hipFree( device_ptr );
	}

	if ( nullptr != intermediary_ptr )
	{
		delete intermediary_ptr;
	}

	if ( nullptr != host_ptr )
	{
		delete host_ptr;
	}
}

void GraphShared::host2device()
{
	const std::size_t N = host().get.number_of_nodes();
	const std::size_t M = host().get.offset( N );

	hipMemcpy( intermediary().set.adjmat(),	host().get.adjmat(),	sizeof( std::uint8_t ) * N * N,	hipMemcpyHostToDevice );
	hipMemcpy( intermediary().set.adjlist(),	host().get.adjlist(),	sizeof( std::int32_t ) * M,		hipMemcpyHostToDevice );
	hipMemcpy( intermediary().set.degrees(),	host().get.degrees(),	sizeof( std::int32_t ) * N,		hipMemcpyHostToDevice );
	hipMemcpy( intermediary().set.offsets(),	host().get.offsets(),	sizeof( uint32_t ) * ( N + 1 ),	hipMemcpyHostToDevice );
}

void GraphShared::device2host()
{
	const std::size_t N = host().get.number_of_nodes();
	const std::size_t M = host().get.offset( N );

	hipMemcpy( host().set.adjmat(),	intermediary().get.adjmat(),	sizeof( std::uint8_t ) * N * N,	hipMemcpyDeviceToHost );
	hipMemcpy( host().set.adjlist(),	intermediary().get.adjlist(),	sizeof( std::int32_t ) * M,		hipMemcpyDeviceToHost );
	hipMemcpy( host().set.degrees(),	intermediary().get.degrees(),	sizeof( std::int32_t ) * N,		hipMemcpyDeviceToHost );
	hipMemcpy( host().set.offsets(),	intermediary().get.offsets(),	sizeof( uint32_t ) * ( N + 1 ),	hipMemcpyDeviceToHost );
}