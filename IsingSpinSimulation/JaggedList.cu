#include "DataStructures.h"
#include <numeric>

using namespace kspace;

template <class elem_type>
JaggedList<elem_type>::JaggedList( const size_t N, const size_t* lengths, const MemoryLocation memloc )
{
	size_t* tmpoffsets = new size_t[ N + 1 ]();
	std::partial_sum( lengths, lengths + N, tmpoffsets + 1 );

	if ( memloc == MemorLocation::host )
	{
		_memloc = new MemoryLocation();
		_data = new elem_type[ tmpoffsets[ N ] ]();
		_length = new size_t();
		_lengths = new size_t[ N ]();
		_offsets = new size_t[ N + 1 ]();

		( *_memloc ) = memloc;
		( *_length ) = N;
		memcpy( _lengths, lengths, sizeof( size_t )*N );
		memcpy( _offsets, tmpoffsets, sizeof( size_t )*( N + 1 ) );
	}
	else if ( memloc == MemoryLocation::device )
	{
		HANDLE_ERROR( hipMalloc( (void**) &_memloc, sizeof( MemoryLocation ) ) );
		HANDLE_ERROR( hipMalloc( (void**) &_data, sizeof( elem_type )*( tmpoffsets[ N ] ) ) );
		HANDLE_ERROR( hipMalloc( (void**) &_length, sizeof( size_t ) ) );
		HANDLE_ERROR( hipMalloc( (void**) &_lengths, sizeof( size_t )*N ) );
		HANDLE_ERROR( hipMalloc( (void**) &_offsets, sizeof( size_t )*( N + 1 ) ) );

		HANDLE_ERROR( hipMemcpy( _memloc, &memloc, sizeof( MemoryLocation ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemset( _data, 0, sizeof( elem_type ) * tmpoffsets[N] ) );
		HANDLE_ERROR( hipMemcpy( _length, &N, sizeof( size_t ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy( _lengths, lengths, sizeof( size_t ) * N, hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy( _offsets, tmpoffsets, sizeof( size_t ) * (N+1), hipMemcpyHostToDevice ) );
	}

	delete[] tmpoffsets;
}

template <class elem_type>
JaggedList<elem_type>::~JaggedList()
{
	if ( memLoc() == MemorLocation::host )
	{
		delete _memloc;
		delete[] _data;
		delete _length;
		delete[] _lengths;
		delete[] _offsets;
	}
	else if ( memLoc() == MemoryLocation::device )
	{
		HANDLE_ERROR( hipFree( _memloc ) );
		HANDLE_ERROR( hipFree( _data ) );
		HANDLE_ERROR( hipFree( _length ) );
		HANDLE_ERROR( hipFree( _lengths ) );
		HANDLE_ERROR( hipFree( _offsets ) );
	}
}

template <class elem_type>
CUDA_CALLABLE_MEMBER MemoryLocation JaggedList<elem_type>::memLoc() const
{
	return *_memloc;
}

template <class elem_type>
CUDA_CALLABLE_MEMBER elem_type JaggedList<elem_type>::get( const size_t row, const size_t col ) const
{
	assert( row >= 0 && row < length() && col >= 0 && col < lengths(row) );
	return _data[ offset( row ) + col ];
}

template <class elem_type>
CUDA_CALLABLE_MEMBER void JaggedList<elem_type>::set( const size_t row, const size_t col, const elem_type val)
{
	assert( row >= 0 && row < length() && col >= 0 && col < lengths( row ) );
	_data[ offset( row ) + col ] = val;
}

template <class elem_type>
CUDA_CALLABLE_MEMBER size_t JaggedList<elem_type>::length() const
{
	return *_length;
}

template <class elem_type>
CUDA_CALLABLE_MEMBER size_t JaggedList<elem_type>::size() const
{
	return _offsets[ length() ];
}

template <class elem_type>
CUDA_CALLABLE_MEMBER size_t JaggedList<elem_type>::length( const size_t row ) const
{
	assert( row >= 0 && row < length() );
	return *_lengths[ row ];
}

template <class elem_type>
CUDA_CALLABLE_MEMBER size_t JaggedList<elem_type>::offset( const size_t row ) const
{
	assert( row >= 0 && row < length() );
	return *_offsets[row];
}