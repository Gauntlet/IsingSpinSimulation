#include "DataStructures.h"
#include <numeric>

using namespace kspace;

template <class elem_type>
JaggedList<elem_type>::JaggedList( const uint32_t N, const uint32_t* lengths, const MemoryLocation memloc )
{
	uint32_t* tmpoffsets = new uint32_t[ N + 1 ]();
	std::partial_sum( lengths, lengths + N, tmpoffsets + 1 );

	if ( memloc == MemorLocation::host )
	{
		_memloc = new MemoryLocation();
		_data = new elem_type[ tmpoffsets[ N ] ]();
		_length = new uint32_t();
		_lengths = new uint32_t[ N ]();
		_offsets = new uint32_t[ N + 1 ]();

		( *_memloc ) = memloc;
		( *_length ) = N;
		memcpy( _lengths, lengths, sizeof( uint32_t )*N );
		memcpy( _offsets, tmpoffsets, sizeof( uint32_t )*( N + 1 ) );
	}
	else if ( memloc == MemoryLocation::device )
	{
		HANDLE_ERROR( hipMalloc( (void**) &_memloc, sizeof( MemoryLocation ) ) );
		HANDLE_ERROR( hipMalloc( (void**) &_data, sizeof( elem_type )*( tmpoffsets[ N ] ) ) );
		HANDLE_ERROR( hipMalloc( (void**) &_length, sizeof( uint32_t ) ) );
		HANDLE_ERROR( hipMalloc( (void**) &_lengths, sizeof( uint32_t )*N ) );
		HANDLE_ERROR( hipMalloc( (void**) &_offsets, sizeof( uint32_t )*( N + 1 ) ) );

		HANDLE_ERROR( hipMemcpy( _memloc, &memloc, sizeof( MemoryLocation ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemset( _data, 0, sizeof( elem_type ) * tmpoffsets[N] ) );
		HANDLE_ERROR( hipMemcpy( _length, &N, sizeof( uint32_t ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy( _lengths, lengths, sizeof( uint32_t ) * N, hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy( _offsets, tmpoffsets, sizeof( uint32_t ) * (N+1), hipMemcpyHostToDevice ) );
	}

	delete[] tmpoffsets;
}

template <class elem_type>
JaggedList<elem_type>::~JaggedList()
{
	if ( memLoc() == MemorLocation::host )
	{
		delete _memloc;
		delete[] _data;
		delete _length;
		delete[] _lengths;
		delete[] _offsets;
	}
	else if ( memLoc() == MemoryLocation::device )
	{
		HANDLE_ERROR( hipFree( _memloc ) );
		HANDLE_ERROR( hipFree( _data ) );
		HANDLE_ERROR( hipFree( _length ) );
		HANDLE_ERROR( hipFree( _lengths ) );
		HANDLE_ERROR( hipFree( _offsets ) );
	}
}

template <class elem_type>
CUDA_CALLABLE_MEMBER MemoryLocation JaggedList<elem_type>::memLoc() const
{
	return *_memloc;
}

template <class elem_type>
CUDA_CALLABLE_MEMBER elem_type JaggedList<elem_type>::get( const uint32_t row, const uint32_t col ) const
{
	assert( row >= 0 && row < length() && col >= 0 && col < lengths(row) );
	return _data[ offset( row ) + col ];
}

template <class elem_type>
CUDA_CALLABLE_MEMBER void JaggedList<elem_type>::set( const uint32_t row, const uint32_t col, const elem_type val)
{
	assert( row >= 0 && row < length() && col >= 0 && col < lengths( row ) );
	_data[ offset( row ) + col ] = val;
}

template <class elem_type>
CUDA_CALLABLE_MEMBER uint32_t JaggedList<elem_type>::length() const
{
	return *_length;
}

template <class elem_type>
CUDA_CALLABLE_MEMBER uint32_t JaggedList<elem_type>::size() const
{
	return _offsets[ length() ];
}

template <class elem_type>
CUDA_CALLABLE_MEMBER uint32_t JaggedList<elem_type>::length( const uint32_t row ) const
{
	assert( row >= 0 && row < length() );
	return *_lengths[ row ];
}

template <class elem_type>
CUDA_CALLABLE_MEMBER uint32_t JaggedList<elem_type>::offset( const uint32_t row ) const
{
	assert( row >= 0 && row < length() );
	return *_offsets[row];
}