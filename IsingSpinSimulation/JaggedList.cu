#include "DataStructures.h"
#include <numeric>

using namespace kspace;

template <class elem_type>
JaggedList<elem_type>::JaggedList( const uint32_t N, const uint32_t* lengths, const MemoryLocation memloc )
{
	uint32_t* tmpoffsets = new uint32_t[ N + 1 ]();
	std::partial_sum( lengths, lengths + N, tmpoffsets + 1 );

	if ( memloc == MemorLocation::host )
	{
		_memloc = new MemoryLocation();
		_data = new elem_type[ tmpoffsets[ N ] ]();
		_length = new uint32_t();
		_lengths = new uint32_t[ N ]();
		_offsets = new uint32_t[ N + 1 ]();

		( *_memloc ) = memloc;
		( *_length ) = N;
		memcpy( _lengths, lengths, sizeof( uint32_t )*N );
		memcpy( _offsets, tmpoffsets, sizeof( uint32_t )*( N + 1 ) );
	}
	else if ( memloc == MemoryLocation::device )
	{
		HANDLE_ERROR( hipMalloc( (void**) &_memloc, sizeof( MemoryLocation ) ) );
		HANDLE_ERROR( hipMalloc( (void**) &_data, sizeof( elem_type )*( tmpoffsets[ N ] ) ) );
		HANDLE_ERROR( hipMalloc( (void**) &_length, sizeof( uint32_t ) ) );
		HANDLE_ERROR( hipMalloc( (void**) &_lengths, sizeof( uint32_t )*N ) );
		HANDLE_ERROR( hipMalloc( (void**) &_offsets, sizeof( uint32_t )*( N + 1 ) ) );

		HANDLE_ERROR( hipMemcpy( _memloc, &memloc, sizeof( MemoryLocation ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemset( _data, 0, sizeof( elem_type ) * tmpoffsets[N] ) );
		HANDLE_ERROR( hipMemcpy( _length, &N, sizeof( uint32_t ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy( _lengths, lengths, sizeof( uint32_t ) * N, hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy( _offsets, tmpoffsets, sizeof( uint32_t ) * (N+1), hipMemcpyHostToDevice ) );
	}

	delete[] tmpoffsets;
}

template <class elem_type>
JaggedList<elem_type>::~JaggedList()
{
	if ( memory_location() == MemorLocation::host )
	{
		delete _memloc;
		delete[] _data;
		delete _length;
		delete[] _lengths;
		delete[] _offsets;
	}
	else if ( memory_location() == MemoryLocation::device )
	{
		HANDLE_ERROR( hipFree( _memloc ) );
		HANDLE_ERROR( hipFree( _data ) );
		HANDLE_ERROR( hipFree( _length ) );
		HANDLE_ERROR( hipFree( _lengths ) );
		HANDLE_ERROR( hipFree( _offsets ) );
	}
}

template <class elem_type>
CUDA_CALLABLE_MEMBER MemoryLocation JaggedList<elem_type>::memory_location() const
{
	return *_memloc;
}

template <class elem_type>
CUDA_CALLABLE_MEMBER elem_type JaggedList<elem_type>::get( const uint32_t row, const uint32_t col ) const
{
	if (row < 0 || row >= numOfRows() || col <= 0 || col >= numOfCols())
	{
		throw std::invalid_argument("jagged list indices out of bounds")
	}

	return _data[ offset( row ) + col ];
}

template <class elem_type>
CUDA_CALLABLE_MEMBER void JaggedList<elem_type>::set( const uint32_t row, const uint32_t col, const elem_type val)
{
	if (row < 0 || row >= numOfRows() || col <= 0 || col >= numOfCols())
	{
		throw std::invalid_argument("jagged list indices out of bounds")
	}

	_data[ offset( row ) + col ] = val;
}

template <class elem_type>
CUDA_CALLABLE_MEMBER uint32_t JaggedList<elem_type>::length() const
{
	return *_length;
}

template <class elem_type>
CUDA_CALLABLE_MEMBER uint32_t JaggedList<elem_type>::size() const
{
	return _offsets[ length() ];
}

template <class elem_type>
CUDA_CALLABLE_MEMBER uint32_t JaggedList<elem_type>::length( const uint32_t row ) const
{
	if (row < 0 || row >= numOfRows())
	{
		throw std::invalid_argument("jagged list rows index out of bounds")
	}

	return *_lengths[ row ];
}

template <class elem_type>
CUDA_CALLABLE_MEMBER uint32_t JaggedList<elem_type>::offset( const uint32_t row ) const
{
	if (row < 0 || row >= numOfRows())
	{
		throw std::invalid_argument("jagged list rows index out of bounds")
	}

	return *_offsets[row];
}