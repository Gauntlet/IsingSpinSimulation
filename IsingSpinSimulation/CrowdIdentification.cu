#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include ""
#include <thrust\scan.h>
#include <thrust\device_ptr.h>


#include "CrowdIdentification.h"
#include <algorithm>
#include "DataStructures.h"
#include "ArrayHandle.h"

using namespace kspace;

class HELPER
{
public:
	template <class T>
	CUDA_CALLABLE_MEMBER static size_t kronecker_delta( T const & a, T const & b )
	{
		return (size_t) ( a == b );
	}

	CUDA_CALLABLE_MEMBER static std::int8_t smooth_state( std::int8_t const & spin_state, double const & flip_average, double const & average_state )
	{
		if ( flip_average >= 0.5 )
		{
			return 0;
		}
		else if ( flip_average < 0.5 )
		{
			if ( average_state < ( -1. / 3. ) )
			{
				return -1;
			}
			else if ( average_state >( 1. / 3. ) )
			{
				return 1;
			}
		}

		return spin_state;
	}

	CUDA_CALLABLE_MEMBER static void smoother_helper( std::int32_t const & node, std::uint32_t const & N, std::uint32_t const & T, std::uint32_t const & window_size, std::int8_t const * spin_states, std::int8_t * smoothed_states )
	{
		size_t flip_count = 0;
		size_t average_state = 0;
		double F, A;
		std::int8_t s1, s2;


		for ( size_t t = 0; t < window_size; ++t )
		{
			s1 = *( spin_states + N*t + node );
			s2 = *( spin_states + N*t + N + node );

			flip_count += 1 - HELPER::kronecker_delta( s1, s2 );
			average_state += s1;
		}

		for ( int t = 0; t < T; ++t )
		{
			size_t lb = std::max( t - window_size, (size_t) 0 );
			size_t ub = std::min( t + window_size, T );

			//Subtract the contribution that has left the averaging window.
			if ( lb > 0 )
			{
				s1 = *( spin_states + N*lb - N + node );
				s2 = *( spin_states + N*lb + node );

				flip_count -= 1 - HELPER::kronecker_delta( s1, s2 );
				average_state -= s1;
			}

			//Add the contribution that has entered the averaging window.
			if ( ub < T )
			{
				s1 = *( spin_states + N*ub - N + node );
				s2 = *( spin_states + N*ub + node );

				flip_count += 1 - HELPER::kronecker_delta( s1, s2 );
				average_state += s2;
			}

			F = (double) flip_count / ( ub - lb );
			A = (double) average_state / ( 2 * window_size + 1 );

			*( smoothed_states + N*t + node ) = HELPER::smooth_state( *( spin_states + N*t + node ), F, A );
		}
	}

	//This is BFS that does not use std::queue or vector. Thus it can be used on both the host and device.
	CUDA_CALLABLE_MEMBER static void partitioner( GRAPH::Graph const & graph, std::int8_t const * spin_states, bool* visited, std::uint32_t* queue, std::uint32_t* offsets, std::uint32_t* partitions, std::int32_t & number_of_partitions )
	{
		//Initialise the visited and queue arrays to zero.
		std::memset( visited, 0, graph.get.number_of_nodes() );
		std::memset( queue, 0, graph.get.number_of_nodes() );

		//Initialise the queue getter and setter.
		size_t queue_front( 0 ), queue_back( 0 );

		//Declare the current node (v) and the neighbour (w).
		std::int32_t v, w;

		//Initialise the partition_id at 0.
		std::uint32_t partition_id = 0;

		//Initialise the offset at 0.
		std::uint32_t offset = 0;

		//Iterate through each node.
		for ( int i = 0; i < graph.get.number_of_nodes(); ++i )
		{
			//If a node has not been visited yet start a BFS from that node.
			if ( !visited[ i ] )
			{
				//Label the node with the partition id.
				partitions[ i ] = partition_id;
				//Mark the node as visited.
				visited[ i ] = true;
				//Add the node to queue.
				queue[ queue_back ] = i;
				//Increment the queue setter.
				queue_back++;

				//Iteratre through the queue until the queue getter catches up with setter.
				while ( queue_front != queue_back )
				{
					//Get a node from the queue.
					v = queue[ queue_front ];
					//Increment the offset.
					offset++;

					//Iterate over node v's neighbours.
					for ( size_t k = 0; k < graph.get.degree( v ); ++k )
					{
						w = graph.get.neighbour( v, k );

						//Check if the neighbour (w) has not been visited.
						if ( !visited[ w ] )
						{
							//Check if the neighbour (w) has the same spin state as node v.
							if ( spin_states[ v ] == spin_states[ w ] )
							{
								//Set w's partition_id.
								partitions[ w ] = partition_id;
								//Set w as visited.
								visited[ w ] = true;

								//Add w to the queue.
								queue[ queue_back ] = w;
								//Increment the queue setter.
								queue_back++;
							}
						}
					}

					//Increment the queue getter.
					queue_front++;
				}

				//Increment partition_id.
				partition_id++;

				//Set the offset for the partition_id.
				if ( partition_id <= graph.get.number_of_nodes() )
				{
					offsets[ partition_id ] = offset;
				}
			}
		}

		number_of_partitions = partition_id;
	}

	CUDA_CALLABLE_MEMBER static void linker( std::uint32_t const & number_of_nodes, std::int32_t const & number_of_partitions, std::int8_t const * spin_states, std::uint32_t const * partitions, std::uint32_t const * offsets, std::uint32_t* similarities, std::int32_t* linklist, std::int32_t & unlinked_count )
	{
		uint32_t p1, p2;

		//The linker first calculates the similarities between partitions
		//in time step t-1 and t.
		//It ensures that similarities are calculated for valid pairs.
		//Consequently that means there is an upper bound of "number_of_nodes" possible
		//similarities to calculate.
		//Once all simialrities are computed we then run over all the partition pairs
		//and find the maximum similarities.
		//If the maximum similarity is also greater than 0.5 then this indicates there
		//is a unique link
		//Iterate over each node.

		for ( int n = 0; n < number_of_nodes; ++n )
		{
			//Check whether the spin state at time t is the same at t+1.
			if ( ( spin_states - number_of_nodes )[ n ] == spin_states[ n ] )
			{
				p1 = ( partitions - number_of_nodes )[ n ];
				p2 = partitions[ n ];

				//Iterate over the linklist checking to see if a link between partition ids
				//at t and t+1 have already been linked and creating a link if they have not.
				//The similarity between the two partitions is also incremented.
				for ( int offset = offsets[ p2 ]; offset < offsets[ p2 + 1 ]; ++offset )
				{
					//If the partition of the node stored in the linklist is the same as the 
					//partition p.first increment the similarity between them by 1.
					if ( ( partitions - number_of_nodes )[ linklist[ offset ] ] == p1 )
					{
						similarities[ offset ]++;
						break;
					}
					else if ( number_of_nodes == linklist[ offset ] )
					{
						//If the node id stored in the linklist is null (where null == number_of_nodes)
						//then set the linklist to the current node id and set the similarity to 1.
						linklist[ offset ] = n;
						similarities[ offset ] = 1;
						break;
					}
				}
			}
		}

		std::uint32_t p1size, p2size;
		std::int32_t n;
		bool islinked;

		//Iterate over each partition.
		for ( int P = 0; P < number_of_partitions; ++P )
		{
			//Initialise islinked to false.
			islinked = false;

			//Set the partition being considered at time step t.
			p2 = P;
			//Compute the size of the partition p.first.
			p2size = offsets[ p2 + 1 ] - offsets[ p2 ];

			//Iterate over all of the potential partitions at time step t+1 to link p.first to.
			for ( int offset = offsets[ p2 ]; offset < offsets[ p2 + 1 ]; ++offset )
			{
				//Set the partition being considered at time step t.
				n = linklist[ offset ];
				p1 = ( partitions - number_of_nodes )[ n ];
				//Compute the size the partition p.first.
				p1size = ( offsets - number_of_nodes )[ p1 + 1 ] - ( offsets - number_of_nodes )[ p1 ];

				//Check that the similarity is greater than 0.5.
				//If there is a similarity greater than 0.5 it is unique and thus 
				//any other similarities can be disregarded
				if ( (double) similarities[ offset ] / (double) ( p1size + p2size - similarities[ offset ] ) > 0.5 )
				{
					//We store the node id of a node in the previous time step as when we relabel the partition_ids
					//into spin_cluster_ids we will not have to update the list with new spin_cluster_ids as they 
					//change.
					linklist[ p2 ] = n;

					islinked = true;

					//If a similarity greater than 0.5 is found then we exit the loop as it is unique.
					break;
				}

			}

			//If a link is not found for this partition then set its link as null (== number_of_nodes).
			//At the same time count the number of unlinked partitions.
			if ( !islinked )
			{
				linklist[ offsets[ p2 ] ] = number_of_nodes;
				unlinked_count++;
			}
		}
	}

	CUDA_CALLABLE_MEMBER static void unlinked_linker( std::uint32_t const & number_of_nodes, std::uint32_t const & number_of_partitions, std::int32_t const & unlinked_count, std::int32_t* linklist )
	{
		std::int32_t unlinked_count_tmp = unlinked_count;
		for ( int P = 0; P < number_of_partitions; ++P )
		{
			if ( number_of_nodes == linklist[ P ] )
			{
				//If a partition is not already linked then set a label using a negative value.
				//This will tell the relabeller to use the absolute value of this number as 
				// this partitions spin_cluster_id.
				linklist[ P ] = -unlinked_count_tmp;
				unlinked_count_tmp++;
			}
		}
	}

	CUDA_CALLABLE_MEMBER static void relabeller( std::int32_t const & node, std::uint32_t const & number_of_nodes, std::uint32_t const & wT, std::int32_t const * linklists, std::uint32_t * partitions )
	{
		std::int32_t L;
		std::uint32_t offset;

		//We iterate over the t time steps relabelling the partition_ids
		//to spin_cluster_ids.
		for ( int t = 0; t < wT; ++t )
		{
			offset = number_of_nodes * t;
			L = linklists[ offset + partitions[ offset + node ] ];
			if ( L < 0 )
			{
				//If the link id 'L' is less than 0 this means that it has
				//no link in the previous time step. Which means it is a 
				//new spin cluster, thus relabel the current partition_id with 
				//the absolute of L as the spin_cluster_id.
				partitions[ offset + node ] = std::abs( L );
			}
			else
			{
				//If the link id 'L' is greater than or equal to 0 then it
				//is the node id of a spin_cluster_id in the previous time 
				//step that P is linked to. Thus we relabel the current node's
				//partition_id to that spin_cluster_id.
				partitions[ offset + node ] = partitions[ offset - number_of_nodes + L ];
			}
		}
	}
};

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

Matrix<std::int8_t> CrowdIdentification::HOST::smoothen( Matrix<std::int8_t>& spin_states, size_t const window_size )
{
	const size_t N = spin_states.get.number_of_rows();
	const size_t T = spin_states.get.number_of_columns();

	Matrix<std::int8_t> smoothed_states( T, N, MemoryLocation::host );

	for ( size_t n = 0; n < N; ++n )
	{
		HELPER::smoother_helper( n, N, T, window_size, spin_states.get.data_ptr(), smoothed_states.set.data_ptr() );
	}

	return smoothed_states;
}

Matrix<std::uint32_t> CrowdIdentification::HOST::partition( GRAPH::Graph const & graph, Matrix<std::int8_t>& smooth_spin_states )
{
	std::uint32_t N = graph.get.number_of_nodes();
	std::uint32_t T = smooth_spin_states.get.number_of_columns();

	Matrix<std::uint32_t> partitions( T, N, MemoryLocation::host );
	Matrix<std::int32_t> linklists( T, N, MemoryLocation::host );

	{
		//During each time step partition the graph into spin clusters.
		ArrayHandle<std::uint32_t> offsets( N + 1 );
		ArrayHandle<bool> visited( N );
		ArrayHandle<std::uint32_t> queue( N );
		ArrayHandle<std::uint32_t> similarities( N );

		std::int32_t number_of_partitions;
		std::int32_t unlinked_count, unlinked_count_tmp;

		//During each time step t we compute the partitions in that time step.
		//Then we calculate the similarity of the partitions between t-1 and t.
		//We then create a linklist which points partitions at time step t to a
		//node in the linked partition at time step t-1.
		//This is done so that when we re-label partition ids using spin-cluster ids
		//the linklist will point to a node in the previous time step that has already
		//been relabelled with the correct spin-cluster id.
		//This allows us to "easily" parallelise the relabelling algorithm.

		//Compute the partitions of the 0-th time step using BFS algorithm.
		HELPER::partitioner( graph, smooth_spin_states.get.data_ptr( 0 ), visited.set.data_ptr(), queue.set.data_ptr(), offsets.set.data_ptr(), partitions.set.data_ptr( 0 ), number_of_partitions );

		//Set the number of unlinked during the first time step to
		//the number of partitions. Then each successive unlinked
		//partition will be automatically given a unique spin_cluster_ids
		//by reading this number and incrementing it.
		unlinked_count = number_of_partitions;

		for ( size_t t = 1; t < smooth_spin_states.get.number_of_columns(); ++t )
		{
			//Compute the partitions of the t-th time step using BFS algorithm.
			HELPER::partitioner( graph, smooth_spin_states.get.data_ptr( t ), visited.set.data_ptr(), queue.set.data_ptr(), offsets.set.data_ptr(), partitions.set.data_ptr( t ), number_of_partitions );

			//Compute the links between partitions which will be used to form the spin clusters.
			//Also get the number of unlinked partitions in this time step.
			unlinked_count_tmp = 0;
			HELPER::linker( graph.get.number_of_nodes, number_of_partitions, smooth_spin_states.get.data_ptr( t ), partitions.get.data_ptr( t ), offsets.get.data_ptr(), similarities.set.data_ptr(), linklists.set.data_ptr( t ), unlinked_count_tmp );

			//Iterate over the partitions and if there are any unlinked partitions.
			//Give them a unique spin_cluster_id using unlinked_count.
			if ( unlinked_count_tmp > 0 )
			{
				HELPER::unlinked_linker( graph.get.number_of_nodes(), number_of_partitions, unlinked_count, linklists.set.data_ptr( t ) );
				//Increment unlinked_count by the number of unlinked partitions in this time step.
				unlinked_count += unlinked_count_tmp;
			}
		}
	}

	for ( std::uint32_t node = 0; node < graph.get.number_of_nodes(); ++node )
	{
		//For each node iterate over each time step and relabel its partition_id to a spin_cluster_id.
		HELPER::relabeller( node, N, T, linklists.get.data_ptr(), partitions.set.data_ptr() );
	}

	return partitions;
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void smoothen_device( std::uint32_t const &N, std::uint32_t const &T, std::int8_t const * spin_states, std::uint32_t const & window_size, std::int8_t * smoothed_states )
{
	int n = blockIdx.x * blockDim.x + threadIdx.x;

	if ( n < N )
	{
		HELPER::smoother_helper( n, N, T, window_size, spin_states, smoothed_states );
	}
}

MatrixShared<std::int8_t> CrowdIdentification::DEVICE::smoothen( Matrix<std::int8_t>& spin_states, std::uint32_t const & window_size )
{
	MatrixShared<std::int8_t> smoothed_states( spin_states.get.number_of_columns(), spin_states.get.number_of_rows() );

	smoothen_device( spin_states.get.number_of_rows(), spin_states.get.number_of_columns(), spin_states.get.data_ptr(), window_size, smoothed_states.device().set.data_ptr() );

	return smoothed_states;
}

__global__ void partitioner_device( std::uint32_t const & N, std::uint32_t const & wT, GRAPH::Graph const * graph, std::int8_t const * spin_states, bool* visited, std::uint32_t* queue, std::uint32_t* offsets, std::uint32_t* partitions, std::int32_t * number_of_partitions )
{
	int t = blockDim.x * blockIdx.x + threadIdx.x;

	if ( t < wT )
	{
		HELPER::partitioner( *graph, spin_states + N*t, visited + N*t, queue + N*t, offsets + N*t, partitions + N*t, *( number_of_partitions + t ) );
	}
}

__global__ void linker_device( std::uint32_t const & N, std::uint32_t const & wT, std::int32_t const * number_of_partitions, std::int8_t const * spin_states, std::uint32_t const * partitions, std::uint32_t const * offsets, std::uint32_t* similarities, std::int32_t* linklist, std::int32_t * unlinked_count )
{
	int t = blockDim.x * blockIdx.x + threadIdx.x;

	if ( t < wT )
	{
		HELPER::linker( N, *( number_of_partitions + t ), spin_states + N*t, partitions + N*t, offsets + t, similarities + t, linklist + t, *( unlinked_count + t ) );
	}
}

__global__ void unlinked_linker_device( std::uint32_t const & N, std::uint32_t const & wT, std::int32_t const * number_of_partitions, std::int32_t const * unlinked_count, std::int32_t* linklist )
{
	int t = blockDim.x * blockIdx.x + threadIdx.x;

	if ( t < wT )
	{
		HELPER::unlinked_linker( N, *( number_of_partitions + t ), *( unlinked_count + t ), linklist + N*t );
	}
}

__global__ void relabeller_device( std::uint32_t const & N, std::uint32_t const & wT, std::int32_t const * linklists, std::uint32_t* partitions )
{
	int n = blockDim.x + blockIdx.x + threadIdx.x;

	if ( n < N )
	{
		HELPER::relabeller( n, N, wT, linklists, partitions );
	}
}

MatrixShared<std::uint32_t> CrowdIdentification::DEVICE::partition( GRAPH::GraphShared const & graph, MatrixShared<std::int8_t>& smooth_spin_states, std::uint32_t const & time_block_size )
{
	const std::uint32_t N = smooth_spin_states.host().get.number_of_rows();
	const std::uint32_t T = smooth_spin_states.host().get.number_of_columns();

	MatrixShared<std::uint32_t> partitions( T, N );

	//We create the following matrices with time_block_size columns, to limit the amount memory
	//being allocated on the device.
	//We can then launch the kernels to work on a block of the partitions at a time.
	//This works as the relabeller relies on the previous time step already being
	//labelled with the correct spin_cluster_id or a new spin_cluster_id being assigned
	//already.
	Matrix<bool> visited( time_block_size, N, MemoryLocation::device );
	Matrix<std::uint32_t> queue( time_block_size, N, MemoryLocation::device );
	Matrix<std::int32_t> linklists( time_block_size, N, MemoryLocation::device );
	Matrix<std::uint32_t> offsets( time_block_size, N, MemoryLocation::device );
	Matrix<std::uint32_t> similarities( time_block_size, N, MemoryLocation::device );

	Matrix<std::int32_t> number_of_partitions( T, 1, MemoryLocation::device );
	MatrixShared<std::int32_t> unlinked_count( T, 1 );

	//Compute the partitions of the 0-th time step using BFS algorithm.
	partitioner_device( N, 1, graph.get.device_ptr(), smooth_spin_states.device().get.data_ptr(), visited.set.data_ptr(), queue.set.data_ptr(), offsets.set.data_ptr(), partitions.device().set.data_ptr(), number_of_partitions.set.data_ptr() );
	//Set the number of unlinked partitions in the 0-th time step as the number of nodes (N)
	//as they do not link back to anything.
	hipMemcpy( unlinked_count.device().set.data_ptr(), &N, sizeof( std::int32_t ), hipMemcpyHostToDevice );
	thrust::device_ptr<std::int32_t> d_ptr;

	for ( std::uint32_t lT = 1; lT < T; lT += time_block_size )
	{
		size_t offset = N*lT;

		partitioner_device( N, time_block_size, graph.get.device_ptr(), smooth_spin_states.device().get.data_ptr() + offset, visited.set.data_ptr(), queue.set.data_ptr(), offsets.set.data_ptr(), partitions.device().set.data_ptr() + offset, number_of_partitions.set.data_ptr() + lT );

		linker_device( N, time_block_size, number_of_partitions.get.data_ptr() + lT, smooth_spin_states.device().get.data_ptr() + offset, partitions.device().get.data_ptr() + offset, offsets.get.data_ptr(), similarities.set.data_ptr(), linklists.set.data_ptr(), unlinked_count.device().set.data_ptr() + lT );

		//Use the thrust library to do an in place prefix-sum (cumulative sum).
		//We first cast the raw device pointer to a device_ptr so that the function knows
		//that the data is on the device.
		d_ptr = thrust::device_pointer_cast<std::int32_t>( unlinked_count.device().set.data_ptr() );
		thrust::exclusive_scan( d_ptr + lT - 1, d_ptr + lT + time_block_size, d_ptr + lT - 1 );

		unlinked_linker_device( N, time_block_size, number_of_partitions.get.data_ptr() + lT, unlinked_count.device().get.data_ptr() + lT, linklists.set.data_ptr() );

		relabeller_device( N, time_block_size, linklists.get.data_ptr(), partitions.device().set.data_ptr() + offset );
	}

	return partitions;
}