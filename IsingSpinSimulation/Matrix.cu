#include "DataStructures.h"

using namespace kspace;

template<class elem_type>
void Matrix<elem_type>::initialise( const uint32_t num_of_columns, const uint32_t num_of_rows, const MemoryLocation memloc ) {
	_memloc = memloc;

	const uint32_t tmplength = num_of_columns * num_of_rows;
	if ( memloc == MemoryLocation::host )
	{
		_data = new elem_type[ tmplength ]();
		_length = new uint32_t();
		_number_of_columns = new uint32_t();
		_number_of_rows = new uint32_t();

		( *_length ) = tmplength;
		( *_number_of_columns ) = num_of_columns;
		( *_number_of_rows ) = num_of_rows;
	}
	else if ( memloc == MemroyLoc::device )
	{
		HANDLE_ERROR( hipMalloc( (void**) &_data, sizeof( elem_type ) * tmplength ) );
		HANDLE_ERROR( hipMalloc( (void**) &_length, sizeof( uint32_t ) ) );
		HANDLE_ERROR( hipMalloc( (void**) &_number_of_columns, sizeof( uint32_t ) ) );
		HANDLE_ERROR( hipMalloc( (void**) &_number_of_rows, sizeof( uint32_t ) ) );

		HANDLE_ERROR( hipMemset( _data, 0, sizeof( elem_type ) * tmplength ) );
		HANDLE_ERROR( hipMemcpy( _length, &tmplength, sizeof( uint32_t ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy( _number_of_columns, &num_of_columns, sizeof( uint32_t ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy( _number_of_rows, &num_of_rows, sizeof( uint32_t ), hipMemcpyHostToDevice ) );
	}
}

template<class elem_type> Matrix<elem_type>::Matrix( const uint32_t N, const MemoryLocation memloc )
{
	initialize( N, N, memloc );
}

template<class elem_type> Matrix<elem_type>::Matrix( const uint32_t num_of_columns, const uint32_t num_of_rows, const MemoryLocation memloc )
{
	initialize( num_of_columns, num_of_rows, memloc );
}

template<class elem_type> Matrix<elem_type>::~Matrix()
{
	if ( memory_location() == MemoryLocation::host )
	{
		delete[] _data;
		delete _length;
		delete _number_of_columns;
		delete _number_of_rows;
	}
	else if ( memory_location() == MemroyLoc::device )
	{
		HANDLE_ERROR( hipFree( _data ) );
		HANDLE_ERROR( hipFree( _length ) );
		HANDLE_ERROR( hipFree( _number_of_columns ) );
		HANDLE_ERROR( hipFree( _number_of_rows ) );
	}
}

template<class elem_type>
Matrix<elem_type>::MemoryLocation memory_location() const
{
	return _memloc;
}

template<class elem_type>
elem_type* Matrix<elem_type>::raw_data()
{
	return _data;
}

template<class elem_type>
elem_type* Matrix<elem_type>::raw_data( const std::uint32_t column )
{
	return _data + number_of_rows() * column;
}

template<class elem_type>
elem_type Matrix<elem_type>::get( const uint32_t column, const uint32_t row ) const
{
	if ( row < 0 || row >= number_of_rows() || column <= 0 || column >= number_of_columns() )
	{
		throw std::invalid_argument( "matrix indices out of bounds" )
	}

	return _data[ column * number_of_rows + row ];
}

template<class elem_type>
void Matrix<elem_type>::set( const uint32_t column, const uint32_t row, const elem_type value )
{
	if ( row < 0 || row >= number_of_rows() || column <= 0 || column >= number_of_columns() )
	{
		throw std::invalid_argument( "matrix indices out of bounds" )
	}

	_data[ column * number_of_rows + row ] = value;
}

template<class elem_type>
uint32_t Matrix<elem_type>::length() const
{
	return *_length;
}

template<class elem_type>
uint32_t Matrix<elem_type>::number_of_columns() const
{
	return *_number_of_columns;
}

template<class elem_type>
uint32_t Matrix<elem_type>::number_of_rows() const
{
	return *_number_of_rows;
}