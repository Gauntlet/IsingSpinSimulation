#include "DataStructures.h"

using namespace kspace;

template<class elem_type> Matrix::initialize( const size_t numofcols, const size_t numofrows, const MemoryLocation memloc ) {

	const size_t tmplength = numofcols * numofrows;

	if ( memloc == MemoryLocation::host )
	{
		_memloc = new MemoryLocation();
		_data = new elem_type[ tmplength ]();
		_length = new size_t();
		_numOfCols = new size_t();
		_numOfRows = new size_t();

		( *_memloc ) = memloc;
		( *_length ) = tmplength;
		( *_numOfCols ) = numofcols;
		( *_numOfRows ) = numofrows;
	}
	else if ( memloc == MemroyLoc::device )
	{
		HANDLE_ERROR( hipMalloc( (void**) &_memloc, sizeof( MemoryLocation ) ) );
		HANDLE_ERROR( hipMalloc( (void**) &_data, sizeof( elem_type )*tmplength ) );
		HANDLE_ERROR( hipMalloc( (void**) &_length, sizeof( size_t ) ) );
		HANDLE_ERROR( hipMalloc( (void**) &_numOfCols, sizeof( size_t ) ) );
		HANDLE_ERROR( hipMalloc( (void**) &_numOfRows, sizeof( size_t ) ) );

		HANDLE_ERROR( hipMemcpy( _memlock, &memloc, sizeof( MemoryLocation ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemset( _data, 0, sizeof( elem_type )*tmplength ) );
		HANDLE_ERROR( hipMemcpy( _length, &tmplength, sizeof( size_t ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy( _numOfCols, &numofcols, sizeof( size_t ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy( _numOfRows, &numofrows, sizeof( size_t ), hipMemcpyHostToDevice ) );
	}
}

template<class elem_type> Matrix::Matrix( const size_t N, const MemoryLocation memloc )
{
	initialize( N, N, memloc );
}

template<class elem_type> Matrix::Matrix( const size_t M, const size_t N, const MemoryLocation memloc )
{
	initialize( M, N, memloc );
}

template<class elem_type> Matrix::~Matrix()
{
	if ( memLoc() == MemoryLocation::host )
	{
		delete _memloc;
		delete[] _data;
		delete _length;
		delete _numOfCols;
		delete _numOfRows;
	}
	else if ( memLoc() == MemroyLoc::device )
	{
		HANDLE_ERROR( hipFree( _memloc ) );
		HANDLE_ERROR( hipFree( _data ) );
		HANDLE_ERROR( hipFree( _length ) );
		HANDLE_ERROR( hipFree( _numOfCols ) );
		HANDLE_ERROR( hipFree( _numOfRows ) );
	}
}

template<class elem_type>
CUDA_CALLABLE_MEMBER Matrix::MemoryLocation memLoc() const
{
	return *_memloc;
}

template<class elem_type>
CUDA_CALLABLE_MEMBER elem_type Matrix::get( const size_t row, const size_t col ) const
{
	assert( row >= 0 && row < numOfRows() && col >= 0 && col < numOfCols() );
	return _data[ row * numOfColumns() + col ];
}

template<class elem_type>
CUDA_CALLABLE_MEMBER void Matrix::set( const size_t row, const size_t col, const elem_type value )
{
	assert( row >= 0 && row < numOfRows() && col >= 0 && col < numOfCols() );
	_data[ row * numOfColumns() + col ] = value;
}

template<class elem_type>
CUDA_CALLABLE_MEMBER size_t Matrix::length() const
{
	return *_length;
}

template<class elem_type>
CUDA_CALLABLE_MEMBER size_t Matrix::numOfColumns() const
{
	return *_numOfCols;
}

template<class elem_type>
CUDA_CALLABLE_MEMBER size_t Matrix::numOfRows() const
{
	return *_numOfRows;
}