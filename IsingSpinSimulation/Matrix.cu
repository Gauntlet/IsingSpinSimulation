#include "Matrix.h"

using namespace kspace;


template<class elem_type>
void Matrix<elem_type>::initialise( const uint32_t num_of_columns, const uint32_t num_of_rows, const MemoryLocation memloc )
{
	_memloc = memloc;

	const uint32_t tmplength = num_of_columns * num_of_rows;
	if ( memloc == MemoryLocation::host )
	{
		_data = new elem_type[ tmplength ]();
		_length = new uint32_t();
		_number_of_columns = new uint32_t();
		_number_of_rows = new uint32_t();

		( *_length ) = tmplength;
		( *_number_of_columns ) = num_of_columns;
		( *_number_of_rows ) = num_of_rows;
	}
	else if ( memloc == MemroyLoc::device )
	{
		HANDLE_ERROR( hipMalloc( (void**) &_data, sizeof( elem_type ) * tmplength ) );
		HANDLE_ERROR( hipMalloc( (void**) &_length, sizeof( uint32_t ) ) );
		HANDLE_ERROR( hipMalloc( (void**) &_number_of_columns, sizeof( uint32_t ) ) );
		HANDLE_ERROR( hipMalloc( (void**) &_number_of_rows, sizeof( uint32_t ) ) );

		HANDLE_ERROR( hipMemset( _data, 0, sizeof( elem_type ) * tmplength ) );
		HANDLE_ERROR( hipMemcpy( _length, &tmplength, sizeof( uint32_t ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy( _number_of_columns, &num_of_columns, sizeof( uint32_t ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy( _number_of_rows, &num_of_rows, sizeof( uint32_t ), hipMemcpyHostToDevice ) );
	}
}

template <class elem_type>
void Matrix<elem_type>::move_data( Matrix<elem_type>&& that )
{
	delete[] data;
	delete length;
	delete number_of_columns;
	delete number_of_rows;

	memlock = that.memloc;
	data = that.data;
	length = that.length;
	number_of_columns = that.number_of_columns;
	number_of_rows = that.number_of_rows;

	that.memloc = NULL;
	that.data = nullptr;
	that.length = nullptr;
	that.number_of_columns = nullptr;
	that.number_of_rows = nullptr;
}

template<class elem_type>
void Matrix<elem_type>::Matrix() : get(*this), set(*this), memloc(MemoryLocation::host), data_ptr(nullptr), length(nullptr), number_of_columns(nullptr), number_of_rows(nullptr) {}

template<class elem_type>
Matrix<elem_type>::Matrix( const uint32_t N, const MemoryLocation memloc ) : get( *this ), set( *this )
{
	initialize( N, N, memloc );
}

template<class elem_type>
Matrix<elem_type>::Matrix( const uint32_t num_of_columns, const uint32_t num_of_rows, const MemoryLocation memloc ) : get( *this ), set( *this )
{
	initialize( num_of_columns, num_of_rows, memloc );
}

template<class elem_type>
Matrix<elem_type>::~Matrix()
{
	if ( memory_location() == MemoryLocation::host )
	{
		delete[] _data;
		delete _length;
		delete _number_of_columns;
		delete _number_of_rows;
	}
	else if ( memory_location() == MemroyLoc::device )
	{
		HANDLE_ERROR( hipFree( _data ) );
		HANDLE_ERROR( hipFree( _length ) );
		HANDLE_ERROR( hipFree( _number_of_columns ) );
		HANDLE_ERROR( hipFree( _number_of_rows ) );
	}
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


template <class elem_type>
MemoryLocation const & Matrix<elem_type>::MATRIX_GET::memory_location() const
{
	return parent.memloc;
}

template <class elem_type>
elem_type const & Matrix<elem_type>::MATRIX_GET::operator()( const size_t column, const size_t row ) const
{
	if ( column >= number_of_columns() || row >= number_of_rows() )
	{
		throw std::out_of_range( "Matrix indices out of range." );
	}

	return parent.data_ptr[ column * number_of_rows() + row ];
}

template <class elem_type>
elem_type const * Matrix<elem_type>::MATRIX_GET::data_ptr() const
{
	return parent.data_ptr;
}

template <class elem_type>
elem_type const * Matrix<elem_type>::MATRIX_GET::data_ptr( const std::uint32_t column ) const
{
	if ( column >= number_of_columns() )
	{
		throw std::out_of_range( "Column index is greater than number of columns" );
	}

	return parent.data_ptr + column*number_of_rows();
}

template <class elem_type>
uint32_t const & Matrix<elem_type>::MATRIX_GET::number_of_columns() const
{
	return parent.number_of_columns;
}

template <class elem_type>
uint32_t const & Matrix<elem_type>::MATRIX_GET::number_of_rows() const
{
	return parent.number_of_rows;
}

template <class elem_type>
uint32_t const & Matrix<elem_type>::MATRIX_GET::length() const
{
	return parent.length;
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template <class elem_type>
elem_type& Matrix<elem_type>::MATRIX_SET::operator()( const size_t column, const size_t row ) const
{
	if ( column >= parent.get.number_of_columns() || row >= parent.get.number_of_rows() )
	{
		throw std::out_of_range( "Matrix indices out of range." );
	}

	return parent.data_ptr[ column*parent.get.number_of_rows() + row ];
}

template <class elem_type>
elem_type* Matrix<elem_type>::MATRIX_SET::data_ptr() const
{
	return parent.data_ptr;
}

template <class elem_type>
elem_type* Matrix<elem_type>::MATRIX_SET::data_ptr( const std::uint32_t column ) const
{
	return parent.data_ptr + column*parent.get.number_of_rows();
}