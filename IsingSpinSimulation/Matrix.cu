#include "DataStructures.h"

using namespace kspace;

template<class elem_type> Matrix::initialize( const uint32_t numofcols, const uint32_t numofrows, const MemoryLocation memloc ) {

	const uint32_t tmplength = numofcols * numofrows;

	if ( memloc == MemoryLocation::host )
	{
		_memloc = new MemoryLocation();
		_data = new elem_type[ tmplength ]();
		_length = new uint32_t();
		_numOfCols = new uint32_t();
		_numOfRows = new uint32_t();

		( *_memloc ) = memloc;
		( *_length ) = tmplength;
		( *_numOfCols ) = numofcols;
		( *_numOfRows ) = numofrows;
	}
	else if ( memloc == MemroyLoc::device )
	{
		HANDLE_ERROR( hipMalloc( (void**) &_memloc, sizeof( MemoryLocation ) ) );
		HANDLE_ERROR( hipMalloc( (void**) &_data, sizeof( elem_type )*tmplength ) );
		HANDLE_ERROR( hipMalloc( (void**) &_length, sizeof( uint32_t ) ) );
		HANDLE_ERROR( hipMalloc( (void**) &_numOfCols, sizeof( uint32_t ) ) );
		HANDLE_ERROR( hipMalloc( (void**) &_numOfRows, sizeof( uint32_t ) ) );

		HANDLE_ERROR( hipMemcpy( _memlock, &memloc, sizeof( MemoryLocation ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemset( _data, 0, sizeof( elem_type )*tmplength ) );
		HANDLE_ERROR( hipMemcpy( _length, &tmplength, sizeof( uint32_t ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy( _numOfCols, &numofcols, sizeof( uint32_t ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy( _numOfRows, &numofrows, sizeof( uint32_t ), hipMemcpyHostToDevice ) );
	}
}

template<class elem_type> Matrix::Matrix( const uint32_t N, const MemoryLocation memloc )
{
	initialize( N, N, memloc );
}

template<class elem_type> Matrix::Matrix( const uint32_t M, const uint32_t N, const MemoryLocation memloc )
{
	initialize( M, N, memloc );
}

template<class elem_type> Matrix::~Matrix()
{
	if ( memory_location() == MemoryLocation::host )
	{
		delete _memloc;
		delete[] _data;
		delete _length;
		delete _numOfCols;
		delete _numOfRows;
	}
	else if ( memory_location() == MemroyLoc::device )
	{
		HANDLE_ERROR( hipFree( _memloc ) );
		HANDLE_ERROR( hipFree( _data ) );
		HANDLE_ERROR( hipFree( _length ) );
		HANDLE_ERROR( hipFree( _numOfCols ) );
		HANDLE_ERROR( hipFree( _numOfRows ) );
	}
}

template<class elem_type>
CUDA_CALLABLE_MEMBER Matrix::MemoryLocation memory_location() const
{
	return *_memloc;
}

template<class elem_type>
CUDA_CALLABLE_MEMBER elem_type Matrix::get( const uint32_t row, const uint32_t col ) const
{
	if (row < 0 || row >= numOfRows() || col <= 0 || col >= numOfCols())
	{
		throw std::invalid_argument("matrix indices out of bounds")
	}

	return _data[ row * numOfColumns() + col ];
}

template<class elem_type>
CUDA_CALLABLE_MEMBER void Matrix::set( const uint32_t row, const uint32_t col, const elem_type value )
{
	if (row < 0 || row >= numOfRows() || col <= 0 || col >= numOfCols())
	{
		throw std::invalid_argument("matrix indices out of bounds")
	}

	_data[ row * numOfColumns() + col ] = value;
}

template<class elem_type>
CUDA_CALLABLE_MEMBER uint32_t Matrix::length() const
{
	return *_length;
}

template<class elem_type>
CUDA_CALLABLE_MEMBER uint32_t Matrix::numOfColumns() const
{
	return *_numOfCols;
}

template<class elem_type>
CUDA_CALLABLE_MEMBER uint32_t Matrix::numOfRows() const
{
	return *_numOfRows;
}