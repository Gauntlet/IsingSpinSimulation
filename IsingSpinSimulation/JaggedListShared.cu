#include "DataStructures.h"

using namespace kspace;

template <class elem_type>
JaggedListShared<elem_type>::JaggedListShared( const uint32_t N, const uint32_t* lengths )
{
	host = new JaggedList( N, lengths, MemoryLocation::host );
	intermediary = new JaggetList( N, lengths, MemoryLocation::device );
	HANDLE_ERROR( hipMalloc( (void**) &device, sizeof( JaggedList ) ) );
	HANDLE_ERROR( hipMemcpy( device, intermediary, sizeof( JaggedList ), hipMemcpyHostToDevice ) );

	HANDLE_ERROR( hipMemcpy( intermediary->_memloc, host->_memloc, sizeof( MemoryLocation ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy( intermediary->_data, host->_data, sizeof( elem_type )*host->size(), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy( intermediary->_length, host->_length, sizeof( uint32_t ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy( intermediary->_lengths, host->_lengths, sizeof( uint32_t )*host->length(), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy( intermediary->_offsets, host->_offsets, sizeof( uint32_t )*( host->length() + 1 ), hipMemcpyHostToDevice ) );
}

template <class elem_type>
JaggedListShared::JaggedListShared()
{
	HANDLE_ERROR( hipFree( device ) );
	delete host;
	delete intermediary;
}

template <class elem_type>
void kspace::JaggedListShared<elem_type>::host2device()
{
	HANDLE_ERROR( hipMemcpy( intermediary->_data, host->_data, sizeof( elem_type )*host->size(), hipMemcpyHostToDevice ) );
}

template <class elem_type>
void JaggedListShared<elem_type>::device2host()
{
	HANDLE_ERROR( hipMemcpy( host->_data, intermediary->_data, sizeof( elem_type )*host->size(), hipMemcpyDeviceToHost ) );
}