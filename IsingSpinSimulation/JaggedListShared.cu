#include "DataStructures.h"

using namespace kspace;

template <class elem_type> 
JaggedListShared::JaggedListShared( const size_t N, const size_t* lengths )
{
	host = new JaggedList( N, lengths, MemoryLocation::host );
	intermediary = new JaggetList( N, lengths, MemoryLocation::device );
	hipMalloc( (void**) &device, sizeof( JaggedList ) );
	hipMemcpy( device, intermediary, sizeof( JaggedList ), hipMemcpyHostToDevice );
	
	hipMemcpy( intermediary->_memloc, host->_memloc, sizeof( MemoryLocation ), hipMemcpyHostToDevice );
	hipMemcpy( intermediary->_data, host->_data, sizeof( elem_type )*host->size(), hipMemcpyHostToDevice );
	hipMemcpy( intermediary->_length, host->_length, sizeof( size_t ), hipMemcpyHostToDevice );
	hipMemcpy( intermediary->_lengths, host->_lengths, sizeof( size_t )*host->length(), hipMemcpyHostToDevice );
	hipMemcpy( intermediary->_offsets, host->_offsets, sizeof( size_t )*( host->length() + 1 ), hipMemcpyHostToDevice );
}

template <class elem_type>
JaggedListShared::JaggedListShared()
{
	hipFree(device);
	delete host;
	delete intermediary;
}

template <class elem_type>
void JaggedListShared::host2device()
{
	hipMemcpy( intermediary->_data,	host->_data,	sizeof( elem_type )*host->size(),			hipMemcpyHostToDevice );
}

template <class elem_type>
void JaggedListShared::device2host()
{
	hipMemcpy( host->_data,	intermediary->_data,	sizeof( elem_type )*host->size(),			hipMemcpyDeviceToHost );
}