#include "DataStructures.h"

using namespace kspace;

template <class elem_type> 
JaggedListShared::JaggedListShared( const uint32_t N, const uint32_t* lengths )
{
	host = new JaggedList( N, lengths, MemoryLocation::host );
	intermediary = new JaggetList( N, lengths, MemoryLocation::device );
	hipMalloc( (void**) &device, sizeof( JaggedList ) );
	hipMemcpy( device, intermediary, sizeof( JaggedList ), hipMemcpyHostToDevice );
	
	hipMemcpy( intermediary->_memloc, host->_memloc, sizeof( MemoryLocation ), hipMemcpyHostToDevice );
	hipMemcpy( intermediary->_data, host->_data, sizeof( elem_type )*host->size(), hipMemcpyHostToDevice );
	hipMemcpy( intermediary->_length, host->_length, sizeof( uint32_t ), hipMemcpyHostToDevice );
	hipMemcpy( intermediary->_lengths, host->_lengths, sizeof( uint32_t )*host->length(), hipMemcpyHostToDevice );
	hipMemcpy( intermediary->_offsets, host->_offsets, sizeof( uint32_t )*( host->length() + 1 ), hipMemcpyHostToDevice );
}

template <class elem_type>
JaggedListShared::JaggedListShared()
{
	hipFree(device);
	delete host;
	delete intermediary;
}

template <class elem_type>
void JaggedListShared::host2device()
{
	hipMemcpy( intermediary->_data,	host->_data,	sizeof( elem_type )*host->size(),			hipMemcpyHostToDevice );
}

template <class elem_type>
void JaggedListShared::device2host()
{
	hipMemcpy( host->_data,	intermediary->_data,	sizeof( elem_type )*host->size(),			hipMemcpyDeviceToHost );
}